#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t desWithCuda(unsigned long*, const unsigned long*, const unsigned long*, unsigned int);
__device__ unsigned long des(const unsigned long, const unsigned long);
__device__ unsigned long ip(const unsigned long);

__global__ void desKernel(unsigned long *ciphertext, const unsigned long *plaintext, const unsigned long *k)
{
    int i = threadIdx.x;
	ciphertext[i] = des(plaintext[i], k[i]);
}
/* --------------------------------
   DES-Verschl�sselungsfunktion
   m ist der Klartext (64 Bit)
   k ist der DES-Schl�ssel (64 Bit)
  -------------------------------- */
__device__ unsigned long des(const unsigned long m, const unsigned long k)
{
	unsigned long M;	// 64 Bit Klartext
	unsigned int L;		// 32 Bit Linker Block von M
	unsigned int R;		// 32 Bit Rechter Block von M
	unsigned int C;		// 28 Bit Teilschl�ssel
	unsigned int D;		// 28 Bit Teilschl�ssel

	// 64 Bit DES-Schl�ssel auf zwei Bitfolgen (28 Bit) C und D abbilden
	C = PC1_C(k);
	D = PC1_D(k);

	// 1. Schritt: Initiale Permutation auf m
	M = ip(m);

	// 2. Schritt: M in Linken und Rechten Block splitten
	L = (int) (M >> 32);		// logisch shiften
	R = (int) (M << 32 >> 32);

	// 3. Schritt: 16 DES-Runden
	for (int i = 1; i < 17; i++)
	{
		unsigned long K = makeRoundKey(C, D, i);
		unsigned int newL = R;
		unsigned int newR = f(R,K) ^ L;
		R = newR;
		L = newL;
	}

}

/* --------------------------------------------------------------------------
   Abbildung f
   Bildet den rechten Block R (32 Bit) und den Rundenschl�ssel K (48 Bit)
   auf Bitfolge f(R,K) (32 Bit) ab.
   -------------------------------------------------------------------------- */
__device__ unsigned int f(unsigned int R, unsigned long K)
{
	unsigned long exp_R;

	// Expansionsfunktion E(R), 32 Bit -> 48 Bit
	exp_R = (unsigned long) R;

	// HIER WEITER MACHEN!!!
}

/* --------------------------------
   Abbildung PC1_C
   Bildet 64 Bit DES-Schl�ssel auf 
   Bitfolge C (28 Bit) ab.
   -------------------------------- */
__device__ unsigned int PC1_C(const unsigned long k)
{
	// HIER PC1 PERMUTATION VON 64 Bit k auf 28 Bit C
}

/* --------------------------------
   Abbildung PC1_D
   Bildet 64 Bit DES-Schl�ssel auf 
   Bitfolge D (28 Bit) ab.
   -------------------------------- */
__device__ unsigned int PC1_D(const unsigned long k)
{
	// HIER PC1 PERMUTATION VON 64 Bit k auf 28 Bit D
}

// makeKey erzeugt einen 48 Bit langen Rundenschl�ssel
__device__ unsigned long makeRoundKey(unsigned int C, unsigned int D, int i)
{
	// v(i) ist die Anzahl der zirkul�ren Linksshifts
	unsigned short v;

	if (i == 1 || i == 2 || i == 9 || i == 16)
		v = 1;
	else
		v = 2;

	// C und D um v bitweise rotieren 
	C = ((C << v) | (C >> 28-v)) & 0xFFFFFFF;
	D = ((D << v) | (D >> 28-v)) & 0xFFFFFFF;

	return PC2(C, D);
}

__device__ unsigned long PC2(unsigned int C, unsigned int D)
{
	// HIER DIE PERMUTATION "PC2" auf C
}

__device__ unsigned long ip(const unsigned long m)
{
	// HIER NOCH DIE INITIALE PERMUTATION AUF m DURCHF�HREN!
	return m;
}

int main()
{
    const int arraySize = 3;
    const unsigned long klartext[arraySize] = { 1, 2, 3};
    const unsigned long key[arraySize] = { 3, 2, 1 };
    unsigned long cipher[arraySize] = { 0 };

    // Verschl�sselt die Klartexte parallel.
    hipError_t cudaStatus = desWithCuda(cipher, klartext, key, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    printf("{%d,%d,%d} + {%d,%d,%d} = {%d,%d,%d}\n",
        klartext[0], klartext[1], klartext[2], key[0], key[1], key[0], cipher[0], cipher[1], cipher[2]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t desWithCuda(unsigned long *c, const unsigned long *p, const unsigned long *k, unsigned int size)
{
    unsigned long *klartext = 0;
    unsigned long *key = 0;
    unsigned long *cipher = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&cipher, size * sizeof(unsigned long));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&klartext, size * sizeof(unsigned long));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&key, size * sizeof(unsigned long));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(klartext, p, size * sizeof(unsigned long), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(key, k, size * sizeof(unsigned long), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
	// <<<dimGrid, dimBlock>>>
	desKernel<<<1, size>>>(cipher, klartext, key);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, cipher, size * sizeof(unsigned long), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(cipher);
    hipFree(klartext);
    hipFree(key);
    
    return cudaStatus;
}
